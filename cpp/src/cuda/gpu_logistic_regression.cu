
#include <hip/hip_runtime.h>
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#ifdef CUDA_AND_GPU
#include "include/gpu_logistic_regression.h"
#include "src/cuda/cuda_matrix_vector_multiply_shared_memory.cu"
#include "include/util.h"
#include "include/matrix.h"
#include "include/vector.h"
#include <cmath>
#include <chrono>
#include "include/cuda_matrix_vector_multiply.h"
#include "include/cuda_matrix_vector_multiply_shared_memory.h"
#include "include/gpu_operations.h"
#include "include/gpu_util.h"


using namespace std::chrono;

namespace Nice {
  /// Calculates the hypothesis of a given input Vector
  ///
  /// \param input
  /// Input Vector
  ///
  /// \return
  /// This function returns a Vector of type T
  template <typename T>
  __device__ T h(T input) {
    return 1 / ((exp(-1 * input) + 1));
  }
  /// CUDA kernel for predict functionality
  template <typename T>
  __global__ void PredictKernel(T *d_theta, T *d_inputs, T *d_predictions,
    int input_x, int input_y, T theta_0){
    //extern __shared__ T theta_tile[];
    SharedMemory<T> shared;
    T* theta_tile = shared.getPointer();
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;

    T sum = 0.0f;
    if (col >= input_x) return;

    for (int p = 0; p < std::ceil((T)input_y / (blockDim.x)); p++){
      T * d_input_tile = &d_inputs[(p * blockDim.x * input_x) + (blockDim.x * blockRow)];
      theta_tile[threadCol] = d_theta[blockDim.x * p + threadCol];
      __syncthreads();
      for (int i = 0; i < blockDim.x; i++){
        int xGIndex = p * blockDim.x + i;
        int yGIndex = col;
        if (xGIndex < input_y && yGIndex < input_x){
          sum += d_input_tile[(input_x * i) + threadCol] * theta_tile[i];
        }
      }
    }
    d_predictions[row * input_x + col] = h(sum + theta_0);
  }

 /** // First kernel for fit functionality. Fit is split into two kernels
  template <typename T>
  __global__ void FitKernel(T *d_xin, T *d_y, T *d_theta, T *d_storage,
    int iterations, T alpha, int input_x, int input_y){
    extern __shared__ T theta_tile[];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;

    if (col >= input_x) return;

    float sum = 0.0f;
    if (iterations == 0 && col < input_y){
      d_theta[col] = 0.0;
    }

    // Tiled multiplication of d_xin matrix and theta vector
    for (int p = 0; p < std::ceil((float)input_y / (blockDim.x)); p++){
      T * d_input_tile = &d_xin[(p * blockDim.x * input_x) + (blockDim.x * blockRow)];
      theta_tile[threadCol] = d_theta[blockDim.x * p + threadCol];
      __syncthreads();
      for (int i = 0; i < blockDim.x; i++){
        int xGIndex = p * blockDim.x + i;
        int yGIndex = col;
        if (xGIndex < input_y && yGIndex < input_x){
          sum += d_input_tile[(input_x * i) + threadCol] * theta_tile[i + 1];
        }
      }
    }


    float theta_0 = d_theta[0];
    // Calls sigmoid function on each value of result + theta_0
    __syncthreads();

    // Stores result - d_y to global memory to be passed to helper function
    d_storage[col] =  h(sum + theta_0) - d_y[col];
    __syncthreads();
  }

  template <typename T>
  __global__ void FitKernelHelper(T *d_xin, T *d_y, T *d_theta, T *d_storage,
    int iterations, T alpha, int input_x, int input_y){

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;
    if (col > input_y) return;
    extern __shared__ T gradient[];

    float sum = 0.0f;

    gradient[col] = 0.0;

    // Tiled multiplication of the transpose of d_xin and result from previous kernel
    if (col < input_y){
      T * d_input_tile = &d_xin[(blockRow * blockDim.x * input_x) + (threadCol * input_x)];
      for (int i = 0; i < input_x; i++){
        sum += d_input_tile[i] * d_storage[i];
      }
      __syncthreads();
      // Stores result into gradient
      gradient[col + 1] = sum;
    }
    atomicAdd(&gradient[0], d_theta[col]);
    // Calculates new theta value
    d_theta[col] = d_theta[col] - ((alpha / input_x) * gradient[col]);
  }**/

  template <typename T>
  __global__ void preMultiply(T * d_result, T *d_y, T *d_temp, T theta_0){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    T value = d_result[col] + theta_0;
    d_temp[col] = h(value) - d_y[col];
  }

  template <typename T>
  __global__ void calculateTheta(T *d_gradient, T *d_theta, T factor, int theta_size){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ T theta_0;
    theta_0 = 0;
    if (col < theta_size){
      atomicAdd(&theta_0, d_theta[col]);
      if (col == 0){
        d_theta[0] = d_theta[0] - (factor * theta_0);
      }
      else{
        d_theta[col] = d_theta[col] - (factor * d_gradient[col - 1]);
      }
    }

  }


  /// Given a set of features and parameters creates a vector of target outputs
  ///
  /// \param inputs
  /// Matrix of input conditions
  ///
  /// \param thetas
  /// Vector of parameters to fit with input conditions
  ///
  /// \return
  /// This function returns a Vector of target outputs of type T
  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuPredict(const Matrix<T> &inputs) {
    int m = inputs.rows();
    int k = inputs.cols();
    T theta_0 = theta[0];
    Vector<T> new_theta = (theta.bottomRows(theta.rows()-1));
    const T * h_theta = &new_theta(0);
    const T * h_inputs = &inputs(0);
    Vector<T> h_predictions(m);


    T * d_theta;
    T * d_inputs;
    T * d_predictions;

    // Setup GPU memory
    CUDA_CALL(cudaMalloc(&d_inputs, (m * k) * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_inputs, h_inputs, (m * k) * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_theta, k * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_theta, h_theta, k * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_predictions, m * sizeof(T)));
    CUDA_CALL(cudaMemset(d_predictions, 0, m * sizeof(T)));

    // Launch kernel here
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid(std::ceil((T) m / (BLOCK_SIZE * BLOCK_SIZE)));

    PredictKernel<<<dimGrid, dimBlock, BLOCK_SIZE * BLOCK_SIZE * sizeof(T)>>>(d_theta, d_inputs,
      d_predictions, m, k, theta_0);
    CUDA_CALL(cudaDeviceSynchronize());

    CUDA_CALL(cudaMemcpy(&h_predictions(0), d_predictions, m * sizeof(T),
      cudaMemcpyDeviceToHost));
    CUDA_CALL(cudaFree(d_theta));
    CUDA_CALL(cudaFree(d_inputs));
    CUDA_CALL(cudaFree(d_predictions));
    return h_predictions;
  }
  /**
  /// Generates a set of parameters from a given training set
  ///
  /// \param xin
  /// Matrix of featuresAcademic
  ///
  /// \param y
  /// Vector of target variables for each set of features
  template <typename T>
  void GpuLogisticRegression<T>::GpuFit(const Matrix<T> &xin, const Vector<T> &y,
    int iterations, T alpha){
      int m = xin.rows();
      int k = xin.cols();

      const T * h_xin = &xin(0);
      const T * h_y = &y(0);
      Vector<T> h_theta(k+1);

      T * d_xin;
      T * d_y;
      T * d_theta;
      T * d_storage;

      // Setup GPU memory
      CUDA_CALL(cudaMalloc(&d_xin, (m * k) * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_xin, h_xin, (m * k) * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_y, h_y, m * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_theta, (k + 1) * sizeof(T)));
      CUDA_CALL(cudaMemset(d_theta, 0, (k + 1) * sizeof(T)));

      CUDA_CALL(cudaMalloc(&d_storage, m * sizeof(T)));
      CUDA_CALL(cudaMemset(d_storage, 0, m * sizeof(T)));

      // Launch kernel here
      dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
      dim3 dimGrid(std::ceil((float)m / (BLOCK_SIZE * BLOCK_SIZE)));

      dim3 dimHelperB(BLOCK_SIZE * BLOCK_SIZE);
      dim3 dimHelperG(std::ceil((float)k / (BLOCK_SIZE * BLOCK_SIZE)));


      for (int i = 0; i < iterations; i++) {
        FitKernel<<<dimGrid, dimBlock, BLOCK_SIZE * BLOCK_SIZE * sizeof(T) >>>(d_xin, d_y,
          d_theta, d_storage, i, alpha, m, k);
        FitKernelHelper<<<dimHelperG, dimHelperB, k * sizeof(T) >>>(d_xin, d_y,
          d_theta, d_storage, i, alpha, m, k);
      }
      CUDA_CALL(cudaDeviceSynchronize());

      CUDA_CALL(cudaMemcpy(&h_theta(0), d_theta, (k + 1) * sizeof(T),
        cudaMemcpyDeviceToHost));
      CUDA_CALL(cudaFree(d_theta));
      CUDA_CALL(cudaFree(d_xin));
      CUDA_CALL(cudaFree(d_storage));
      CUDA_CALL(cudaFree(d_y));
      theta = h_theta;
  }
  **/

  /**template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuPredictMV(const Matrix<T> &inputs) {
    Vector<T> predictions, yhat;
    Matrix<T> product;

    CUDA_CALL(cudaMalloc(&d_xin, xin.size() * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_xin, h_xin, xin.size() * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_y, y.size() * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_y, h_y, y.size() * sizeof(T),
      cudaMemcpyHostToDevice));

    product = inputs * theta.bottomRows(theta.rows()-1);
    yhat = product.rowwise().sum();
    yhat = yhat.array() + theta(0);
    predictions = h(yhat);
    return predictions;
  }**/

  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuFitMV(const Matrix<T> &xin, const Vector<T> &y,
    const Matrix<T> &predict_inputs, int iterations, T alpha){
      Vector<T> gradient;
      theta.resize(xin.cols() + 1);
      gradient.resize(theta.rows());
      theta.setZero();
      gradient.setZero();

      Vector<T> bottom_theta = theta.bottomRows(theta.rows() - 1);
      Matrix<T> xin_trans = xin.transpose();
      Vector<T> temp(xin.rows());

      // Fit
      Vector<T> h_result(xin.rows());
      Vector<T> h_gradient(xin.cols() + 1);
      Vector<T> h_theta(xin.cols() + 1);

      const T * h_xin = &xin(0);
      const T * h_y = &y(0);
      const T * h_temp = &h_result(0);
      const T * h_xin_trans = &xin_trans(0);
      const T * h_bottom_theta = &bottom_theta(0);

      // Predict
      Vector<T> h_predictions(predict_inputs.rows());

      const T * h_predict_inputs = &predict_inputs(0);

      // Fit
      T * d_xin;
      T * d_y;
      T * d_xin_trans;
      T * d_bottom_theta;
      T * d_result;
      T * d_gradient;
      T * d_temp;
      T * d_theta;

      // Predict
      T * d_predictions;
      T * d_predict_inputs;

      // Setup GPU memory
      /**CUDA_CALL(cudaMalloc(&d_xin, xin.size() * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_xin, h_xin, xin.size() * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_y, y.size() * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_y, h_y, y.size() * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_xin_trans, xin.size() * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_xin_trans, h_xin_trans, xin.size() * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_predict_inputs, predict_inputs.size() * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_predict_inputs, h_predict_inputs, predict_inputs.size() * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_bottom_theta, bottom_theta.rows() * sizeof(T)));

      CUDA_CALL(cudaMalloc(&d_result, xin.rows() * sizeof(T)));
      CUDA_CALL(cudaMalloc(&d_gradient, xin.size() * sizeof(T)));
      CUDA_CALL(cudaMalloc(&d_temp, xin.size() * sizeof(T)));

      CUDA_CALL(cudaMalloc(&d_theta, theta.size() * sizeof(T)));
      CUDA_CALL(cudaMemset(d_theta, 0, theta.size() * sizeof(T)));

      CUDA_CALL(cudaMalloc(&d_predictions, predict_inputs.rows() * sizeof(T)));**/

      for (int i = 0; i < iterations; i++) {
        bottom_theta = theta.bottomRows(theta.rows() - 1);
        /**CUDA_CALL(cudaMemcpy(d_bottom_theta, h_bottom_theta, bottom_theta.rows() * sizeof(T),
          cudaMemcpyHostToDevice));
        CUDA_CALL(cudaMemset(d_result, 0, xin.rows() * sizeof(T)));

        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(std::ceil((T)xin.rows()/ (BLOCK_SIZE)));

        CudaSharedMVKernel<<<dimGrid, dimBlock, BLOCK_SIZE * sizeof(T)>>>
          (d_xin, d_bottom_theta, d_result, xin.rows(), xin.cols());

        CUDA_CALL(cudaMemcpy(&h_result(0), d_result, xin.rows() * sizeof(T),
          cudaMemcpyDeviceToHost));**/
        Nice::CudaSharedMVMultiply<T> global_op(1024);
        h_result = global_op.Multiply(xin, bottom_theta);
        // h_result = xin * bottom_theta;
        h_result = h_result.array() + theta(0);
        gradient.bottomRows(gradient.rows() - 1) =
          global_op.Multiply(xin.transpose(), (h(h_result) - y));
          // xin.transpose() * (h(h_result) - y);
        gradient(0) = theta.sum();
        theta = theta - ((alpha/ y.size()) * gradient);

        /**CUDA_CALL(cudaMemset(d_temp, 0, xin.rows() * sizeof(T)));
        CUDA_CALL(cudaMemcpy(d_temp, h_temp, xin.rows() * sizeof(T),
          cudaMemcpyHostToDevice));

        preMultiply<<<dimGrid, dimBlock>>>(d_result, d_y, d_temp, theta(0));

        CUDA_CALL(cudaMemset(d_gradient, 0, xin.rows() * sizeof(T)));
        dim3 dimGrid2(std::ceil((float)xin_trans.rows()/ (BLOCK_SIZE)));

        CudaSharedMVKernel<<<dimGrid2, dimBlock, BLOCK_SIZE * sizeof(T)>>>
          (d_xin_trans, d_temp, d_gradient, xin.cols(), xin.rows());

        dim3 dimGrid3(std::ceil((float)theta.size()/ (BLOCK_SIZE)));
        calculateTheta<<<dimGrid3, dimBlock, sizeof(T)>>>(d_gradient, d_theta, alpha / y.size(), theta.size());
        CUDA_CALL(cudaMemcpy(&theta(0), d_theta, theta.size() * sizeof(T),
          cudaMemcpyDeviceToHost));

        CUDA_CALL(cudaMalloc(&d_theta, theta.size() * sizeof(T)));
        CUDA_CALL(cudaMemcpy(d_theta, &theta(0), theta.size() * sizeof(T),
          cudaMemcpyHostToDevice));**/

        Vector<T> yhat;
        Matrix<T> product;
        product = predict_inputs * theta.bottomRows(theta.rows()-1);
        yhat = product.rowwise().sum();
        yhat = yhat.array() + theta(0);
        h_predictions = h(yhat);
        h_predictions = h_predictions.unaryExpr(std::ptr_fun<T,T>(std::round));
      }

      CUDA_CALL(cudaDeviceSynchronize());

      CUDA_CALL(cudaFree(d_xin));
      CUDA_CALL(cudaFree(d_y));
      CUDA_CALL(cudaFree(d_xin_trans));
      CUDA_CALL(cudaFree(d_bottom_theta));
      CUDA_CALL(cudaFree(d_result));
      CUDA_CALL(cudaFree(d_gradient));
      CUDA_CALL(cudaFree(d_temp));
      CUDA_CALL(cudaFree(d_theta));
      CUDA_CALL(cudaFree(d_predictions));
      CUDA_CALL(cudaFree(d_predict_inputs));

      // CUDA_CALL(cudaDeviceSynchronize());
      //
      // // high_resolution_clock::time_point t2 = high_resolution_clock::now();
      // // auto duration = duration_cast<microseconds>( t2 - t1 ).count();
      // // std::cout << "CUDA Logistic Regression - Fit: " << (long)duration << std::endl;
      //
      // CUDA_CALL(cudaMemcpy(&h_theta(0), d_theta, (k + 1) * sizeof(T),
      //   cudaMemcpyDeviceToHost));
      // CUDA_CALL(cudaFree(d_theta));
      // CUDA_CALL(cudaFree(d_xin));
      // CUDA_CALL(cudaFree(d_storage));
      // CUDA_CALL(cudaFree(d_y));
      // theta = h_theta;
      return h_predictions;
  }

  /**template
  void GpuLogisticRegression<float>::GpuFit(const Matrix<float> &xin, const Vector<float> &y,
    int iterations, float alpha);**/

  template
  Vector<float> GpuLogisticRegression<float>::GpuFitMV(const Matrix<float> &xin, const Vector<float> &y,
      const Matrix<float> &predict_inputs, int iterations, float alpha);

  template
  Vector<float> GpuLogisticRegression<float>::GpuPredict(const Matrix<float> &inputs);

  template
  Vector<double> GpuLogisticRegression<double>::GpuFitMV(const Matrix<double> &xin, const Vector<double> &y,
      const Matrix<double> &predict_inputs, int iterations, double alpha);

  template
  Vector<double> GpuLogisticRegression<double>::GpuPredict(const Matrix<double> &inputs);


}; // namespace Nice
#endif  //CUDA_AND_GPU
