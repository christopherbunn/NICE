
#include <hip/hip_runtime.h>
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#ifdef CUDA_AND_GPU
#include "include/gpu_logistic_regression.h"
#include <cmath>
#include <chrono>

using namespace std::chrono;

namespace Nice {
  /// Calculates the hypothesis of a given input Vector
  ///
  /// \param input
  /// Input Vector
  ///
  /// \return
  /// This function returns a Vector of type T
  template <typename T>
  __device__ T h(T input) {
    return 1 / ((exp(-1 * input) + 1));
  }


  /// CUDA kernel for predict functionality
  template <typename T>
  __global__ void PredictKernel(T *d_theta, T *d_inputs, T *d_predictions,
    int input_x, int input_y, T theta_0){
    extern __shared__ T theta_tile[];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;

    float sum = 0.0f;
    if (col >= input_x) return;

    for (int p = 0; p < std::ceil((float)input_y / (blockDim.x)); p++){
      T * d_input_tile = &d_inputs[(p * blockDim.x * input_x) + (blockDim.x * blockRow)];
      theta_tile[threadCol] = d_theta[blockDim.x * p + threadCol];
      __syncthreads();
      for (int i = 0; i < blockDim.x; i++){
        int xGIndex = p * blockDim.x + i;
        int yGIndex = col;
        if (xGIndex < input_y && yGIndex < input_x){
          sum += d_input_tile[(input_x * i) + threadCol] * theta_tile[i];
        }
      }
    }
    d_predictions[row * input_x + col] = h(sum + theta_0);
  }

  // First kernel for fit functionality. Fit is split into two kernels
  template <typename T>
  __global__ void FitKernel(T *d_xin, T *d_y, T *d_theta, T *d_storage,
    int iterations, T alpha, int input_x, int input_y){
    extern __shared__ T theta_tile[];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;

    if (col >= input_x) return;

    float sum = 0.0f;
    if (iterations == 0 && col < input_y){
      d_theta[col] = 0.0;
    }

    // Tiled multiplication of d_xin matrix and theta vector
    for (int p = 0; p < std::ceil((float)input_y / (blockDim.x)); p++){
      T * d_input_tile = &d_xin[(p * blockDim.x * input_x) + (blockDim.x * blockRow)];
      theta_tile[threadCol] = d_theta[blockDim.x * p + threadCol];
      __syncthreads();
      for (int i = 0; i < blockDim.x; i++){
        int xGIndex = p * blockDim.x + i;
        int yGIndex = col;
        if (xGIndex < input_y && yGIndex < input_x){
          sum += d_input_tile[(input_x * i) + threadCol] * theta_tile[i + 1];
        }
      }
    }


    float theta_0 = d_theta[0];
    // Calls sigmoid function on each value of result + theta_0
    float temp = h(sum + theta_0);
    __syncthreads();

    // Stores result - d_y to global memory to be passed to helper function
    d_storage[col] =  temp - d_y[col];
    __syncthreads();
  }

  template <typename T>
  __global__ void FitKernelHelper(T *d_xin, T *d_y, T *d_theta, T *d_storage,
    int iterations, T alpha, int input_x, int input_y){

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;
    if (col > input_y) return;
    extern __shared__ T gradient[];

    float sum = 0.0f;

    gradient[col] = 0.0;

    // Tiled multiplication of the transpose of d_xin and result from previous kernel
    if (col < input_y){
      T * d_input_tile = &d_xin[(blockRow * blockDim.x * input_x) + (threadCol * input_x)];
      for (int i = 0; i < input_x; i++){
        sum += d_input_tile[i] * d_storage[i];
      }
      __syncthreads();
      // Stores result into gradient
      gradient[col + 1] = sum;
    }
    atomicAdd(&gradient[0], d_theta[col]);
    // Calculates new theta value
    d_theta[col] = d_theta[col] - ((alpha / input_x) * gradient[col]);
  }


  /// Given a set of features and parameters creates a vector of target outputs
  ///
  /// \param inputs
  /// Matrix of input conditions
  ///
  /// \param thetas
  /// Vector of parameters to fit with input conditions
  ///
  /// \return
  /// This function returns a Vector of target outputs of type T
  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuPredict(const Matrix<T> &inputs) {
    int m = inputs.rows();
    int k = inputs.cols();
    T theta_0 = theta[0];
    Vector<T> new_theta = (theta.bottomRows(theta.rows()-1));
    const T * h_theta = &new_theta(0);
    const T * h_inputs = &inputs(0);
    Vector<T> h_predictions(m);


    T * d_theta;
    T * d_inputs;
    T * d_predictions;

    // Setup GPU memory
    CUDA_CALL(cudaMalloc(&d_inputs, (m * k) * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_inputs, h_inputs, (m * k) * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_theta, k * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_theta, h_theta, k * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_predictions, m * sizeof(T)));
    CUDA_CALL(cudaMemset(d_predictions, 0, m * sizeof(T)));

    // Launch kernel here
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid(std::ceil((float) m / (BLOCK_SIZE * BLOCK_SIZE)));

    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    PredictKernel<<<dimGrid, dimBlock, BLOCK_SIZE * BLOCK_SIZE * sizeof(T)>>>(d_theta, d_inputs,
      d_predictions, m, k, theta_0);
    CUDA_CALL(cudaDeviceSynchronize());

    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>( t2 - t1 ).count();
    std::cout << "CUDA Logistic Regression - Predict: " << (long)duration << std::endl;

    CUDA_CALL(cudaMemcpy(&h_predictions(0), d_predictions, m * sizeof(T),
      cudaMemcpyDeviceToHost));
    CUDA_CALL(cudaFree(d_theta));
    CUDA_CALL(cudaFree(d_inputs));
    CUDA_CALL(cudaFree(d_predictions));
    return h_predictions;
  }

  /// Generates a set of parameters from a given training set
  ///
  /// \param xin
  /// Matrix of featuresAcademic
  ///
  /// \param y
  /// Vector of target variables for each set of features
  template <typename T>
  void GpuLogisticRegression<T>::GpuFit(const Matrix<T> &xin, const Vector<T> &y,
    int iterations, T alpha){
      int m = xin.rows();
      int k = xin.cols();

      const T * h_xin = &xin(0);
      const T * h_y = &y(0);
      Vector<T> h_theta(k+1);

      T * d_xin;
      T * d_y;
      T * d_theta;
      T * d_storage;

      // Setup GPU memory
      CUDA_CALL(cudaMalloc(&d_xin, (m * k) * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_xin, h_xin, (m * k) * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_y, h_y, m * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_theta, (k + 1) * sizeof(T)));
      CUDA_CALL(cudaMemset(d_theta, 0, (k + 1) * sizeof(T)));

      CUDA_CALL(cudaMalloc(&d_storage, m * sizeof(T)));
      CUDA_CALL(cudaMemset(d_storage, 0, m * sizeof(T)));

      // Launch kernel here
      dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
      dim3 dimGrid(std::ceil((float)m / (BLOCK_SIZE * BLOCK_SIZE)));

      dim3 dimHelperB(BLOCK_SIZE * BLOCK_SIZE);
      dim3 dimHelperG(std::ceil((float)k / (BLOCK_SIZE * BLOCK_SIZE)));

      high_resolution_clock::time_point t1 = high_resolution_clock::now();

      for (int i = 0; i < iterations; i++) {
        FitKernel<<<dimGrid, dimBlock, BLOCK_SIZE * BLOCK_SIZE * sizeof(T) >>>(d_xin, d_y,
          d_theta, d_storage, i, alpha, m, k);
        FitKernelHelper<<<dimHelperG, dimHelperB, k * sizeof(T) >>>(d_xin, d_y,
          d_theta, d_storage, i, alpha, m, k);
      }
      CUDA_CALL(cudaDeviceSynchronize());

      high_resolution_clock::time_point t2 = high_resolution_clock::now();
      auto duration = duration_cast<microseconds>( t2 - t1 ).count();
      std::cout << "CUDA Logistic Regression - Fit: " << (long)duration << std::endl;

      CUDA_CALL(cudaMemcpy(&h_theta(0), d_theta, (k + 1) * sizeof(T),
        cudaMemcpyDeviceToHost));
      CUDA_CALL(cudaFree(d_theta));
      CUDA_CALL(cudaFree(d_xin));
      CUDA_CALL(cudaFree(d_storage));
      CUDA_CALL(cudaFree(d_y));
      theta = h_theta;
  }

  template
  void GpuLogisticRegression<float>::GpuFit(const Matrix<float> &xin, const Vector<float> &y,
    int iterations, float alpha);

  template
  Vector<float> GpuLogisticRegression<float>::GpuPredict(const Matrix<float> &inputs);


}; // namespace Nice
#endif  //CUDA_AND_GPU
