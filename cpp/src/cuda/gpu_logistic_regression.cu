
#include <hip/hip_runtime.h>
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#ifdef CUDA_AND_GPU
#include "include/gpu_logistic_regression.h"
#include "include/util.h"
#include "include/matrix.h"
#include "include/vector.h"
#include <cmath>
#include <chrono>
#include "include/gpu_operations.h"
#include "include/gpu_util.h"
#define BLOCK_SIZE 32


using namespace std::chrono;

namespace Nice {
  // Used to be able to use templates with shared memory
  template <>
  struct SharedMemory <float>
  {
      __device__ float *getPointer()
      {
          extern __shared__ float s_float[];
          return s_float;
      }
  };

  // Used to be able to use templates with shared memory
  template <>
  struct SharedMemory <double>
  {
      __device__ double *getPointer()
      {
          extern __shared__ double s_double[];
          return s_double;
      }
  };

  /// Calculates the hypothesis of a given input Vector
  ///
  /// \param input
  /// Input Vector
  ///
  /// \return
  /// This function returns a Vector of type T
  template <typename T>
  __device__ T h(T input) {
    return 1 / ((exp(-1 * input) + 1));
  }
  /// CUDA kernel for predict functionality
  template <typename T>
  __global__ void PredictKernel(T *d_theta, T *d_inputs, T *d_predictions,
    int input_x, int input_y, T theta_0){
    //extern __shared__ T theta_tile[];
    SharedMemory<T> shared;
    T* theta_tile = shared.getPointer();
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;

    T sum = 0.0f;
    if (col >= input_x) return;

    for (int p = 0; p < std::ceil((T)input_y / (blockDim.x)); p++){
      T * d_input_tile = &d_inputs[(p * blockDim.x * input_x) + (blockDim.x * blockRow)];
      theta_tile[threadCol] = d_theta[blockDim.x * p + threadCol];
      __syncthreads();
      for (int i = 0; i < blockDim.x; i++){
        int xGIndex = p * blockDim.x + i;
        int yGIndex = col;
        if (xGIndex < input_y && yGIndex < input_x){
          sum += d_input_tile[(input_x * i) + threadCol] * theta_tile[i];
        }
      }
    }
    d_predictions[row * input_x + col] = h(sum + theta_0);
  }

  template <typename T>
  __global__ void preMultiply(T * d_result, T *d_y, T *d_temp, T *d_theta){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    T value = d_result[col] + d_theta[0];
    d_temp[col] = h(value) - d_y[col];
  }

  /// Given a set of features and parameters creates a vector of target outputs
  ///
  /// \param inputs
  /// Matrix of input conditions
  ///
  /// \param thetas
  /// Vector of parameters to fit with input conditions
  ///
  /// \return
  /// This function returns a Vector of target outputs of type T
  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuPredict(const Matrix<T> &inputs) {
    int m = inputs.rows();
    int k = inputs.cols();
    T theta_0 = theta[0];
    Vector<T> new_theta = (theta.bottomRows(theta.rows()-1));
    const T * h_theta = &new_theta(0);
    const T * h_inputs = &inputs(0);
    Vector<T> h_predictions(m);


    T * d_theta;
    T * d_inputs;
    T * d_predictions;

    // Setup GPU memory
    CUDA_CALL(cudaMalloc(&d_inputs, (m * k) * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_inputs, h_inputs, (m * k) * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_theta, k * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_theta, h_theta, k * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_predictions, m * sizeof(T)));
    CUDA_CALL(cudaMemset(d_predictions, 0, m * sizeof(T)));

    // Launch kernel here
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid(std::ceil((T) m / (BLOCK_SIZE * BLOCK_SIZE)));

    PredictKernel<<<dimGrid, dimBlock, BLOCK_SIZE * BLOCK_SIZE * sizeof(T)>>>(d_theta, d_inputs,
      d_predictions, m, k, theta_0);
    CUDA_CALL(cudaDeviceSynchronize());

    CUDA_CALL(cudaMemcpy(&h_predictions(0), d_predictions, m * sizeof(T),
      cudaMemcpyDeviceToHost));
    CUDA_CALL(cudaFree(d_theta));
    CUDA_CALL(cudaFree(d_inputs));
    CUDA_CALL(cudaFree(d_predictions));
    return h_predictions;
  }

  template <typename T>
  __global__ void CudaSharedKernel(T *d_a, T *d_x, T *d_y, int const a_rows, int const x_size) {
    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;
    SharedMemory<T> shared;
    T* xTile = shared.getPointer();

    __syncthreads();
    T sum = 0.0f;
    for (int p = 0; p < std::ceil((T)x_size / (BLOCK_SIZE)); p++){
      for (int i = 0; i < BLOCK_SIZE; i++){
        T * aTile = &d_a[(p * BLOCK_SIZE * a_rows) + (BLOCK_SIZE * blockRow)];
        xTile[threadCol] = d_x[BLOCK_SIZE * p + threadCol];
        int xGIndex = p * BLOCK_SIZE + i;
        int yGIndex = blockIdx.x * BLOCK_SIZE + threadIdx.x;
        if (xGIndex < x_size && yGIndex < a_rows){
          sum += aTile[(a_rows *i) + threadCol] * xTile[i];
        }
      }
    }
    __syncthreads();
    d_y[threadCol + (blockRow * BLOCK_SIZE)] = sum;
  }

    template <typename T>
    __global__ void CudaGlobalKernel(T *d_a, T *d_x, T *d_y, int a_rows, int x_size) {
      int row = blockIdx.y * blockDim.y + threadIdx.y;
      int col = blockIdx.x * blockDim.x + threadIdx.x;
      T sum = 0.0f;
      if (row >= x_size || col >= a_rows) return;
      for (int k = 0; k < x_size; k++) {
        sum += (d_a[col + (k * a_rows)] * d_x[k]);
      }
      d_y[row * a_rows + col] = sum;
    }

  template <typename T>
   __global__ void reduce(T *in_vector, T *sum, int n){
    SharedMemory<T> shared;
    T *shared_data = shared.getPointer();
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    T end_sum = (i < n) ? in_vector[i] : 0;

    if (i + blockDim.x < n) end_sum += in_vector[i + blockDim.x];
    shared_data[threadIdx.x] = end_sum;

    for (int pos = blockDim.x / 2; pos>0; pos >>= 1)
    {
        if (threadIdx.x < pos){
            shared_data[threadIdx.x] = end_sum = end_sum +
              shared_data[threadIdx.x + pos];
        }
    }
    if (threadIdx.x == 0){
      sum[blockIdx.x] = end_sum;
    }
}

template <typename T>
__global__ void calculateTheta(T *d_gradient, T *d_theta, T factor, int theta_size){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < theta_size){
    d_theta[col] = d_theta[col] - (factor * d_gradient[col]);
  }
}

  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuFit(const Matrix<T> &xin, const Vector<T> &y,
    const Matrix<T> &predict_inputs, int iterations, T alpha){
      Vector<T> gradient;
      theta.resize(xin.cols() + 1);
      gradient.resize(theta.rows());
      theta.setZero();
      gradient.setZero();

      Matrix<T> xin_trans = xin.transpose();
      Vector<T> temp(xin.rows());

      // Fit
      Vector<T> h_result(xin.rows());
      Vector<T> h_end(xin.cols());
      Vector<T> h_temp(xin.rows());
      Vector<T> h_gradient(xin.cols() + 1);
      Vector<T> h_theta(xin.cols() + 1);

      const T * h_xin = &xin(0);
      const T * h_y = &y(0);
      const T * h_xin_trans = &xin_trans(0);

      T * d_xin;
      CUDA_CALL(cudaMalloc(&d_xin, (xin.rows() * xin.cols() * sizeof(T))));
      CUDA_CALL(cudaMemcpy(d_xin, h_xin, (xin.rows() * xin.cols() * sizeof(T)),
        cudaMemcpyHostToDevice));

      T * d_y;
      CUDA_CALL(cudaMalloc(&d_y, y.size() * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_y, h_y, y.size() * sizeof(T),
        cudaMemcpyHostToDevice));

      T * d_xin_trans;
      CUDA_CALL(cudaMalloc(&d_xin_trans, xin_trans.size() * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_xin_trans, h_xin_trans, xin_trans.size() * sizeof(T),
        cudaMemcpyHostToDevice));

      T * d_theta;

      T * d_temp;
      CUDA_CALL(cudaMalloc(&d_temp, xin.rows() * sizeof(T)));
      CUDA_CALL(cudaMemset(d_temp, 0, xin.rows() * sizeof(T)));

      T * d_result;
      CUDA_CALL(cudaMalloc(&d_result, xin.rows() * sizeof(T)));
      CUDA_CALL(cudaMemset(d_result, 0, xin.rows() * sizeof(T)));

      T * d_end;
      CUDA_CALL(cudaMalloc(&d_end, (xin.cols() + 1) * sizeof(T)));
      CUDA_CALL(cudaMemset(d_end, 0, (xin.cols() + 1) * sizeof(T)));

      dim3 dimBlock(BLOCK_SIZE);
      dim3 dimGrid(std::ceil((T)xin.rows() / (BLOCK_SIZE)));

      dim3 dimBlockTrans(BLOCK_SIZE);
      dim3 dimGridTrans(std::ceil((T)(xin.cols() - 1) / (BLOCK_SIZE)));

      CUDA_CALL(cudaMalloc(&d_theta, theta.size() * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_theta, &theta(0), theta.size() * sizeof(T),
        cudaMemcpyHostToDevice));

      for (int i = 0; i < iterations; i++) {
        CUDA_CALL(cudaDeviceSynchronize());
        CudaSharedKernel<<<dimGrid, dimBlock, BLOCK_SIZE * sizeof(T)>>>(d_xin, d_theta + 1, d_result, xin.rows(), xin.cols());
        CUDA_CALL(cudaDeviceSynchronize());
        preMultiply<<<dimGrid,dimBlock>>>(d_result, d_y, d_temp, d_theta);
        reduce<<< dimGrid, dimBlock, BLOCK_SIZE * sizeof(T)>>>(d_theta, d_end, theta.size());
        CUDA_CALL(cudaDeviceSynchronize());
        CudaSharedKernel<<<dimGridTrans, dimBlockTrans, BLOCK_SIZE * sizeof(T)>>>(d_xin_trans, d_temp, d_end + 1, xin.cols(), xin.rows());
        CUDA_CALL(cudaDeviceSynchronize());
        calculateTheta<<< dimGrid, dimBlock>>>(d_end, d_theta, alpha/ y.size(), theta.size());
      }

      CUDA_CALL(cudaMemcpy(&theta(0), d_theta, theta.size() * sizeof(T), cudaMemcpyDeviceToHost));

      CUDA_CALL(cudaFree(d_xin));
      CUDA_CALL(cudaFree(d_y));
      CUDA_CALL(cudaFree(d_xin_trans));
      CUDA_CALL(cudaFree(d_theta));
      CUDA_CALL(cudaFree(d_temp));
      CUDA_CALL(cudaFree(d_result));
      CUDA_CALL(cudaFree(d_end));
      // Predict
      Vector<T> h_predictions(predict_inputs.rows());

      Vector<T> yhat;
      Matrix<T> product;
      product = predict_inputs * theta.bottomRows(theta.rows()-1);
      yhat = product.rowwise().sum();
      yhat = yhat.array() + theta(0);
      h_predictions = h(yhat);
      h_predictions = h_predictions.unaryExpr(std::ptr_fun<T,T>(std::round));

      return h_predictions;
  }

  template
  Vector<float> GpuLogisticRegression<float>::GpuFit(const Matrix<float> &xin, const Vector<float> &y,
      const Matrix<float> &predict_inputs, int iterations, float alpha);

  template
  Vector<float> GpuLogisticRegression<float>::GpuPredict(const Matrix<float> &inputs);

  template
  Vector<double> GpuLogisticRegression<double>::GpuFit(const Matrix<double> &xin, const Vector<double> &y,
      const Matrix<double> &predict_inputs, int iterations, double alpha);

  template
  Vector<double> GpuLogisticRegression<double>::GpuPredict(const Matrix<double> &inputs);


}; // namespace Nice
#endif  //CUDA_AND_GPU
