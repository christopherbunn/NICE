
#include <hip/hip_runtime.h>
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#ifdef CUDA_AND_GPU
#include "include/gpu_logistic_regression.h"
#include <cmath>
#include <chrono>

using namespace std::chrono;

namespace Nice {
  /// Calculates the hypothesis of a given input Vector
  ///
  /// \param input
  /// Input Vector
  ///
  /// \return
  /// This function returns a Vector of type T
  template <typename T>
  __device__ T h(T input) {
    return 1 / ((exp(-1 * input) + 1));
  }

  /// CUDA kernel for predict functionality
  template <typename T>
  __global__ void PredictKernel(T *d_theta, T *d_inputs, T *d_predictions,
      int input_x, int input_y, T theta_0){
    extern __shared__ float yhat[];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (col >= input_x) return;
    for (int k = 0; k < input_y; k++) {
      sum += (d_inputs[k * input_x + col] * d_theta[row * input_x + k]);
    }
    __syncthreads();
    yhat[row * input_x + col] = sum + theta_0;
    d_predictions[row * input_x + col] = h(yhat[row * input_x + col]);
    __syncthreads();
  }

  /// Work in progress CUDA kernel for Fit functionality
  template <typename T>
  __global__ void FitKernel(T *d_xin, T *d_y, T *d_theta, T* d_trans,
    int iterations, T alpha, int input_x, int input_y){
    extern __shared__ float shared[];
    // Variables are hard coded for development only
    T * theta = (T*)shared;
    T * gradient = (T*)&theta[(input_y + 1)];
    T * new_theta = (T*)&gradient[input_y + 1];
    T * temp = (T*)&new_theta[input_x];

    // Corresponding row/col variables
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= input_y || col >= input_x) return;
    if (col < input_x + 1){
      theta[row * input_x + col] = 0.0;
    }



    // iterations loop for Fit kernel. The two is hard-coded for testing
    // purposes, it will be replaced by the iterations variable
    for (int i = 0; i < 1; i++) {
      //printf("Col: %i :: %5.5f\n", col, temp[col]);
      float sum = 0.0f;
      if (col < input_y + 1){
        gradient[row * input_x + col] = 0.0;
      }
      if (col < input_x){
        new_theta[row * input_x + col] = 0.0;
        temp[row * input_x + col] = 0.0;
      }

      for (int j = 0; j < input_y; j++) {
        sum += (d_xin[j * input_x + col] * theta[j + 1]);
      }
      __syncthreads();
      new_theta[row * input_x + col] = sum;

      // Adds the value of theta(0) to every value of new_theta
      new_theta[row * input_x + col] = theta[0] +
        new_theta[row * input_x + col];

      // Generates hypothesis from new_theta and subtracts them from y values
      temp[row * input_x + col] = h(new_theta[row * input_x + col]) -
        d_y[row * input_x + col];

      for (int j = 0; j < (input_y); j++) {
        float num = (d_xin[(row+j) * input_x + col] * temp[row * input_x + col]);
        atomicAdd(&gradient[j + 1], num);
      }

      if(col < input_y + 1){
        atomicAdd(&gradient[0], theta[col]);
      }
      __syncthreads();

      /// Sets thetas according to gradient descent equation.
      if (col < input_y + 1){
        theta[row * input_x + col] = theta[row * input_x + col] -
          ((alpha / input_x) * gradient[row * input_x + col]);
      }
      __syncthreads();
    }
    if (col < input_y + 1){
      d_theta[row * input_x + col] = theta[row * input_x + col];
    }
  }

  /// Given a set of features and parameters creates a vector of target outputs
  ///
  /// \param inputs
  /// Matrix of input conditions
  ///
  /// \param thetas
  /// Vector of parameters to fit with input conditions
  ///
  /// \return
  /// This function returns a Vector of target outputs of type T
  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuPredict(const Matrix<T> &inputs) {
    int m = inputs.rows();
    int k = inputs.cols();
    T theta_0 = theta[0];
    Vector<T> new_theta = (theta.bottomRows(theta.rows()-1));
    const T * h_theta = &new_theta(0);
    const T * h_inputs = &inputs(0);
    Vector<T> h_predictions(m);


    T * d_theta;
    T * d_inputs;
    T * d_predictions;

    // Setup GPU memory
    CUDA_CALL(cudaMalloc(&d_inputs, (m * k) * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_inputs, h_inputs, (m * k) * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_theta, k * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_theta, h_theta, k * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_predictions, m * sizeof(T)));
    CUDA_CALL(cudaMemset(d_predictions, 0, m * sizeof(T)));


    // Launch kernel here
    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid(std::ceil((float)m / (BLOCK_SIZE)));

    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    PredictKernel<<<dimGrid, dimBlock, m * sizeof(T)>>>(d_theta, d_inputs,
      d_predictions, m, k, theta_0);
    CUDA_CALL(cudaDeviceSynchronize());
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>( t2 - t1 ).count();
    std::cout << "CUDA Logistic Regression - Predict: " << (long)duration << std::endl;

    CUDA_CALL(cudaMemcpy(&h_predictions(0), d_predictions, m * sizeof(T),
      cudaMemcpyDeviceToHost));
    CUDA_CALL(cudaFree(d_theta));
    CUDA_CALL(cudaFree(d_inputs));
    CUDA_CALL(cudaFree(d_predictions));
    return h_predictions;
  }

  template <typename T>
  Vector<T> h(Vector<T> input) {
    input = ((-1 * input).array().exp()) + 1;
    return input.array().inverse();
  }
  /// Generates a set of parameters from a given training set
  ///
  /// \param xin
  /// Matrix of featuresAcademic
  ///
  /// \param y
  /// Vector of target variables for each set of features
  template <typename T>
  void GpuLogisticRegression<T>::GpuFit(const Matrix<T> &xin, const Vector<T> &y,
    int iterations, T alpha){
      int m = xin.rows();
      int k = xin.cols();

      const T * h_xin = &xin(0);
      const T * h_y = &y(0);
      Vector<T> h_theta(k+1);
      Matrix<T> trans = xin.transpose();
      const T * h_trans = &trans(0);

      T * d_trans;
      T * d_xin;
      T * d_y;
      T * d_theta;

      // Setup GPU memory
      CUDA_CALL(cudaMalloc(&d_xin, (m * k) * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_xin, h_xin, (m * k) * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_y, h_y, m * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_theta, (k + 1) * sizeof(T)));
      CUDA_CALL(cudaMemset(d_theta, 0, (k + 1) * sizeof(T)));

      CUDA_CALL(cudaMalloc(&d_trans, m * k * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_trans, h_trans, (m * k) * sizeof(T),
        cudaMemcpyHostToDevice));

      // Launch kernel here
      dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
      //dim3 dimGrid(std::ceil((float)m / (BLOCK_SIZE)), std::ceil((float)k / (BLOCK_SIZE)));
      FitKernel<<<1, dimBlock, ((3 * m ) + (k+1))>>>(d_xin, d_y,
        d_theta, d_trans, iterations, alpha, m, k);

      CUDA_CALL(cudaDeviceSynchronize());

      CUDA_CALL(cudaMemcpy(&h_theta(0), d_theta, (k + 1) * sizeof(T),
        cudaMemcpyDeviceToHost));
      CUDA_CALL(cudaFree(d_theta));
      CUDA_CALL(cudaFree(d_xin));
      CUDA_CALL(cudaFree(d_y));
      theta = h_theta;
  }

  template
  void GpuLogisticRegression<float>::GpuFit(const Matrix<float> &xin, const Vector<float> &y,
    int iterations, float alpha);

  template
  Vector<float> GpuLogisticRegression<float>::GpuPredict(const Matrix<float> &inputs);


}; // namespace Nice
#endif  //CUDA_AND_GPU
