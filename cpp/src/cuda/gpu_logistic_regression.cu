
#include <hip/hip_runtime.h>
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#ifdef CUDA_AND_GPU
#include "include/gpu_logistic_regression.h"
#include <cmath>

#define BLOCK_DIM 16
#define BLOCK_SIZE 16
namespace Nice {

  /// Calculates the hypothesis of a given input Vector
  ///
  /// \param input
  /// Input Vector
  ///
  /// \return
  /// This function returns a Vector of type T
  template <typename T>
  __device__ T h(T input) {
    return 1 / ((exp(-1 * input) + 1));
  }

  /// CPU Transpose. Used for testing
  template <typename T>
  __device__ T transpose(T * input) {
    return 1 / ((exp(-1 * input) + 1));
  }

  /// CUDA kernel for predict functionality
  template <typename T>
  __global__ void PredictKernel(T *d_theta, T *d_inputs, T *d_predictions, int input_x, int input_y, T theta_0){
    extern __shared__ float yhat[];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row >= input_y || col >= input_x) return;
    for (int k = 0; k < input_y; k++) {
      sum += (d_inputs[k * input_x + col] * d_theta[row * input_x + k]);

    }
    __syncthreads();
    yhat[row * input_x + col] = sum + theta_0;
    d_predictions[row * input_x + col] = h(yhat[row * input_x + col]);
    __syncthreads();
  }

  /// Work in progress CUDA kernel for Fit functionality
  template <typename T>
  __global__ void FitKernel(T *d_xin, T *d_y, T *d_theta, int iterations,
    T alpha, int input_x, int input_y){
    extern __shared__ float theta[];
    // Variables are hard coded for development only
    __shared__ float gradient[3];
    __shared__ float new_theta[3];
    __shared__ float temp[10];

    // Corresponding row/col variables
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= input_y || col >= input_x) return;

    theta[row * input_x + col] = 0.0;
    gradient[row * input_x + col] = 0.0;
    new_theta[row * input_x + col] = 0.0;
    temp[row * input_x + col] = 0.0;

    // iterations loop for Fit kernel. The two is hard-coded for testing
    // purposes, it will be replaced by the iterations variable
    for (int i = 0; i < 2; i++) {

      // Multiplies xin array by current thetas to generate new thetas
      float sum = 0.0f;
      for (int j = 0; j < input_y; j++) {
        sum += (d_xin[j * input_x + col] * theta[row * input_x + (j+ 1)]);
      }
      __syncthreads();
      new_theta[row * input_x + col] = sum;

      // Adds the value of theta(0) to every value of new_theta
      new_theta[row * input_x + col] = theta[row * input_x] +
        new_theta[row * input_x + col];
      __syncthreads();

      // Generates hypothesis from new_theta and subtracts them from y values
      temp[row * input_x + col] = h(new_theta[row * input_x + col]) - d_y[row * input_x + col];

      /// TODO fix transpose functionality
      /// For this function, it is supposed to multiply the transpose of xin by temp.
      /// Currently, it prints out the correct multiplication values but it does not add them together
      /// The current print out shows only the first values of num in the gradient array.
      sum = 0.0f;
      for (int j = 0; j < (input_y); j++) {
          __syncthreads();
          float num = (d_xin[(row+j) * input_x + col] * temp[row * input_x + col]);
          printf("%i: %5.5f * %5.5f = %5.5f\n", j, d_xin[(row+j) * input_x + col], temp[row * input_x + col], d_xin[(row+j) * input_x + col] * temp[row * input_x + col]);
          __syncthreads();
          gradient[j+1] = gradient[j+1] + num;
          __syncthreads();
      }

      /// Sums up theta and sets it to gradient[0]
      for (int j = 1; j < input_x + 1; j++){
        sum += theta[row * input_x];
      }
      __syncthreads();
      gradient[0] = sum;

      /// Sets thetas according to gradient descent equation. 
      __syncthreads();
      d_theta[row * input_x + col] = d_theta[row * input_x + col] -
        ((alpha / input_x) * gradient[row * input_x + col]);
    }
  }
  /// Given a set of features and parameters creates a vector of target outputs
  ///
  /// \param inputs
  /// Matrix of input conditions
  ///
  /// \param thetas
  /// Vector of parameters to fit with input conditions
  ///
  /// \return
  /// This function returns a Vector of target outputs of type T
  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuPredict(const Matrix<T> &inputs, const Vector<T> &theta) {
    int m = inputs.rows();
    int k = inputs.cols();
    T theta_0 = theta[0];
    Vector<T> new_theta = (theta.bottomRows(theta.rows()-1));
    const T * h_theta = &new_theta(0);
    const T * h_inputs = &inputs(0);
    Vector<T> h_predictions(m);

    T * d_theta;
    T * d_inputs;
    T * d_predictions;

    // Setup GPU memory
    CUDA_CALL(cudaMalloc(&d_inputs, (m * k) * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_inputs, h_inputs, (m * k) * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_theta, k * sizeof(T)));
    CUDA_CALL(cudaMemcpy(d_theta, h_theta, k * sizeof(T),
      cudaMemcpyHostToDevice));

    CUDA_CALL(cudaMalloc(&d_predictions, m * sizeof(T)));
    CUDA_CALL(cudaMemset(d_predictions, 0, m * sizeof(T)));
    // Launch kernel here
    dim3 dimBlock(BLOCK_SIZE *BLOCK_SIZE);
    dim3 dimGrid(inputs.rows() * inputs.cols());
    //std::cout <<  (inputs.cols() / dimBlock.x) * (inputs.rows() / dimBlock.y) << "\n";
    PredictKernel<<<dimGrid, dimBlock, (k + 1)>>>(d_theta, d_inputs,
      d_predictions, m, k, theta_0);
    CUDA_CALL(cudaDeviceSynchronize());

    CUDA_CALL(cudaMemcpy(&h_predictions(0), d_predictions, m * sizeof(T),
      cudaMemcpyDeviceToHost));
    CUDA_CALL(cudaFree(d_theta));
    CUDA_CALL(cudaFree(d_inputs));
    CUDA_CALL(cudaFree(d_predictions));
    return h_predictions;
  }

  template <typename T>
  Vector<T> h(Vector<T> input) {
    input = ((-1 * input).array().exp()) + 1;
    return input.array().inverse();
  }
  /// Generates a set of parameters from a given training set
  ///
  /// \param xin
  /// Matrix of featuresAcademic
  ///
  /// \param y
  /// Vector of target variables for each set of features
  template <typename T>
  Vector<T> GpuLogisticRegression<T>::GpuFit(const Matrix<T> &xin, const Vector<T> &y,
    int iterations, T alpha){
      int m = xin.rows();
      int k = xin.cols();

      const T * h_xin = &xin(0);
      const T * h_y = &y(0);
      Vector<T> h_theta(m);

      T * d_xin;
      T * d_y;
      T * d_theta;

      // Setup GPU memory
      CUDA_CALL(cudaMalloc(&d_xin, (m * k) * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_xin, h_xin, (m * k) * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(cudaMemcpy(d_y, h_y, m * sizeof(T),
        cudaMemcpyHostToDevice));

      CUDA_CALL(cudaMalloc(&d_theta, k * sizeof(T)));
      CUDA_CALL(cudaMemset(d_theta, 0, k * sizeof(T)));

      // Launch kernel here
      dim3 dimBlock(BLOCK_SIZE *BLOCK_SIZE);
      dim3 dimGrid(xin.rows() * xin.cols());
      //std::cout <<  (inputs.cols() / dimBlock.x) * (inputs.rows() / dimBlock.y) << "\n";
      FitKernel<<<dimGrid, dimBlock, m>>>(d_xin, d_y,
        d_theta, iterations, alpha, m, k);

      CUDA_CALL(cudaDeviceSynchronize());

      CUDA_CALL(cudaMemcpy(&h_theta(0), d_theta, k * sizeof(T),
        cudaMemcpyDeviceToHost));
      CUDA_CALL(cudaFree(d_theta));
      CUDA_CALL(cudaFree(d_xin));
      CUDA_CALL(cudaFree(d_y));
      return h_theta;
  }

  template
  Vector<float> GpuLogisticRegression<float>::GpuFit(const Matrix<float> &xin, const Vector<float> &y,
    int iterations, float alpha);

  template
  Vector<float> GpuLogisticRegression<float>::GpuPredict(const Matrix<float> &inputs, const Vector<float> &theta);


}; // namespace Nice
#endif  //CUDA_AND_GPU
