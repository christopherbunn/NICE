#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#include "include/cuda_matrix_vector_multiply.h"
#include <chrono>

using namespace std::chrono;

namespace Nice {

  template <typename T>
  __global__ void CudaMatrixVectorMulKernel(T *d_a, T *d_x, T *d_y, int a_rows, int x_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row >= x_size || col >= a_rows) return;
    for (int k = 0; k < x_size; k++) {
      sum += (d_a[k * a_rows + col] * d_x[row * a_rows + k]);
    }
    __syncthreads();
    d_y[row * a_rows + col] = sum;
  }

  template <typename T>
  Vector<T> CudaMatrixVectorMultiply<T>::Multiply(const Matrix<T> &a, const Vector<T> &b) {
    if (a.cols() == b.rows() && !a.isZero()) {
      // Allocate and transfer memories
      int m = a.rows();
      int n = b.cols();
      int k = a.cols();

      const T * h_a = &a(0);
      const T * h_x = &b(0);
      Vector<T> h_y(m);

      T * d_a;
      T * d_x;
      T * d_y;

      // Setup GPU memory
      CUDA_CALL(hipMalloc(&d_a, (m * k) * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_a, h_a, (m * k) * sizeof(T),
        hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_x, k * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_x, h_x, k * sizeof(T),
          hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(hipMemset(d_y, 0, m * sizeof(T)));

      // Launch kernel here

      high_resolution_clock::time_point t1 = high_resolution_clock::now();
      CudaMatrixVectorMulKernel<<<m, 256>>>(d_a, d_x, d_y, m, k);

      // Device sync
      CUDA_CALL(hipDeviceSynchronize());
      high_resolution_clock::time_point t2 = high_resolution_clock::now();
      auto duration = duration_cast<microseconds>( t2 - t1 ).count();
      std::cout << "CUDA global time: " << (long)duration << std::endl;

      // Transfer memories back, clear memrory, and return result
      CUDA_CALL(hipMemcpy(&h_y(0), d_y, m * sizeof(T),
        hipMemcpyDeviceToHost));
      CUDA_CALL(hipFree(d_a));
      CUDA_CALL(hipFree(d_x));
      CUDA_CALL(hipFree(d_y));

      return h_y;
    } else if (a.cols() != b.rows()) {
      std::cerr << "Matricies in gpu matrix multiply's sizes aren't compatible"
                << std::endl;
      exit(1);
    } else if (a.isZero() && b.isZero()) {
      std::cerr << "The maxtrix and the vector are empty"
                << std::endl;
      exit(1);
    } else if (a.isZero()) {
      std::cerr << "The maxtrix is empty"
                << std::endl;
      exit(1);
    } else if (b.isZero()) {
      std::cerr << "The vector is empty"
                << std::endl;
      exit(1);
    } else {
      std::cerr << "Unknown error"
                << std::endl;
      exit(1);
    }
  }
  template
  Vector<float> CudaMatrixVectorMultiply<float>::Multiply(const Matrix<float> &a, const Vector<float> &b);

  template
  Vector<double> CudaMatrixVectorMultiply<double>::Multiply(const Matrix<double> &a, const Vector<double> &b);

}
