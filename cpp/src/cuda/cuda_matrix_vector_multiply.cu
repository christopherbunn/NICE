#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#include "include/cuda_matrix_vector_multiply.h"
#define BLOCK_SIZE 32
namespace Nice {

  template <typename T>
  __global__ void CudaKernel(T *d_a, T *d_x, T *d_y, int a_rows, int x_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    T sum = 0.0f;
    if (row >= x_size || col >= a_rows) return;
    for (int k = 0; k < x_size; k++) {
      sum += (d_a[col + (k * a_rows)] * d_x[k]);
      //printf("k: %i %3.3f * %3.3f  sum: %i r:%i, c:%i x:%i, y:%i \n", k, d_a[col + (k * a_rows)], d_x[k], threadIdx.x, row, col, col, k);
    }
    d_y[row * a_rows + col] = sum;
  }

  template <typename T>
  Vector<T> CudaMatrixVectorMultiply<T>::Multiply(const Matrix<T> &a, const Vector<T> &b) {
    if (a.cols() == b.rows() && !a.isZero()) {
      // Allocate and transfer memories
      int m = a.rows();
      int n = b.cols();
      int k = a.cols();

      const T * h_a = &a(0);
      const T * h_x = &b(0);
      Vector<T> h_y(m);

      T * d_a;
      T * d_x;
      T * d_y;
      //std::cout << a << std::endl;
      //std::cout << b << std::endl;
      // Setup GPU memory
      CUDA_CALL(hipMalloc(&d_a, (m * k) * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_a, h_a, (m * k) * sizeof(T),
        hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_x, k * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_x, h_x, k * sizeof(T),
          hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(hipMemset(d_y, 0, m * sizeof(T)));

      // Launch kernel here
      dim3 dimBlock(BLOCK_SIZE);
      dim3 dimGrid(std::ceil((float)m / (BLOCK_SIZE)));
      CudaKernel<<<dimGrid, dimBlock>>>(d_a, d_x, d_y, m, k);

      // Device sync
      CUDA_CALL(hipDeviceSynchronize());

      // Transfer memories back, clear memrory, and return result
      CUDA_CALL(hipMemcpy(&h_y(0), d_y, m * sizeof(T),
        hipMemcpyDeviceToHost));
      CUDA_CALL(hipFree(d_a));
      CUDA_CALL(hipFree(d_x));
      CUDA_CALL(hipFree(d_y));

      return h_y;
    } else if (a.cols() != b.rows()) {
      std::cerr << "Matricies in gpu matrix multiply's sizes aren't compatible"
                << std::endl;
      exit(1);
    } else if (a.isZero() && b.isZero()) {
      std::cerr << "The maxtrix and the vector are empty"
                << std::endl;
      exit(1);
    } else if (a.isZero()) {
      std::cerr << "The maxtrix is empty"
                << std::endl;
      exit(1);
    } else if (b.isZero()) {
      std::cerr << "The vector is empty"
                << std::endl;
      exit(1);
    } else {
      std::cerr << "Unknown error"
                << std::endl;
      exit(1);
    }
  }
  template
  Vector<float> CudaMatrixVectorMultiply<float>::Multiply(const Matrix<float> &a, const Vector<float> &b);

  template
  Vector<double> CudaMatrixVectorMultiply<double>::Multiply(const Matrix<double> &a, const Vector<double> &b);

}
