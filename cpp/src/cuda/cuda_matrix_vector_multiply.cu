#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#include "include/cuda_matrix_vector_multiply.h"

namespace Nice {
  template <typename T>
  __global__ void CudaMatrixVectorMulKernel(T *d_a, T *d_x, T *d_y, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    for (int k = 0; k < size; k++) {
      sum += d_a[row * size + k] * d_x[k * size + col];
    }
    d_y[row * size + col] = sum;
  }

  template <typename T>

  Vector<T> CudaMatrixVectorMultiply<T>::Multiply(const Matrix<T> &a, const Vector<T> &b) {
    if (a.cols() == b.rows() && !a.isZero()) {
      // Allocate and transfer memories
      int m = a.rows();
      int n = b.cols();
      int k = a.cols();

      const T * h_a = &a(0);
      const T * h_x = &b(0);
      Vector<T> h_y(m);

      T * d_a;
      T * d_x;
      T * d_y;

      // Setup GPU memory
      CUDA_CALL(hipMalloc(&d_a, m * k * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_a, h_a, m * k * sizeof(T),
        hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_x, m * k * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_x, h_x, k * n * sizeof(T),
          hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(hipMemset(d_y, 0, m * sizeof(T)));


      unsigned int blocks = (a.cols() + 255) / 256;
      unsigned int size = a.cols();
      // Launch kernel here
      CudaMatrixVectorMultiply::CudaMatrixVectorMulKernel<<(blocks, 256)>>(d_a, d_x, d_y, size);

      // Device sync
      CUDA_CALL(hipDeviceSynchronize());

      // Transfer memories back, clear memrory, and return result
      CUDA_CALL(hipMemcpy(h_y(0), d_y, m * sizeof(T),
        hipMemcpyDeviceToHost));
      CUDA_CALL(hipFree(d_a));
      CUDA_CALL(hipFree(d_x));

      //util_->SyncMem(d_a, nullptr, 0, false);
      //util_->SyncMem(d_x, nullptr, 0, false);
      //util_->SyncMem(d_y, &h_y(0), m);

      return h_y;
    } else if (a.cols() != b.rows()) {
      std::cerr << "Matricies in gpu matrix multiply's sizes aren't compatible"
                << std::endl;
      exit(1);
    } else if (a.isZero() && b.isZero()) {
      std::cerr << "The maxtrix and the vector are empty"
                << std::endl;
      exit(1);
    } else if (a.isZero()) {
      std::cerr << "The maxtrix is empty"
                << std::endl;
      exit(1);
    } else if (b.isZero()) {
      std::cerr << "The vector is empty"
                << std::endl;
      exit(1);
    } else {
      std::cerr << "Unknown error"
                << std::endl;
      exit(1);
    }
  }
  //template
  //Vector<float> CudaMatrixVectorMultiply<float>::Multiply(const Matrix<float> &a, const Vector<float> &b);

}
