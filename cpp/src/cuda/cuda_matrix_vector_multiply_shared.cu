#include "hip/hip_runtime.h"
// The MIT License (MIT)
//
// Copyright (c) 2016 Northeastern University
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
#include "include/cuda_matrix_vector_multiply_shared_memory.h"
#define BLOCK_SIZE 32

using namespace std::chrono;

namespace Nice {

  template <typename T>
  __global__ void CudaSharedMVKernel(T *d_a, T *d_x, T *d_y, int const a_rows, int const x_size) {
    int blockRow = blockIdx.x;
    int threadCol = threadIdx.x;

    extern __shared__ T xTile[];

    __syncthreads();
    T sum = 0.0f;
    for (int p = 0; p < std::ceil((float)x_size / (BLOCK_SIZE)); p++){
      for (int i = 0; i < BLOCK_SIZE; i++){
        T * aTile = &d_a[(p * BLOCK_SIZE * a_rows) + (BLOCK_SIZE * blockRow)];
        extern __shared__ T xTile[];
        xTile[threadCol] = d_x[BLOCK_SIZE * p + threadCol];
        int xGIndex = p * BLOCK_SIZE + i;
        int yGIndex = blockIdx.x * BLOCK_SIZE + threadIdx.x;
        if (xGIndex < x_size && yGIndex < a_rows){
          sum += aTile[(a_rows *i) + threadCol] * xTile[i];
        }
      }
    }
    __syncthreads();
    d_y[threadCol + (blockRow * BLOCK_SIZE)] += sum;
  }

  template <typename T>
  Vector<T> CudaSharedMVMultiply<T>::Multiply(const Matrix<T> &a, const Vector<T> &b) {
    if (a.cols() == b.rows() && !a.isZero()) {
      int m = a.rows();
      int n = b.cols();
      int k = a.cols();

      const T * h_a = &a(0);
      const T * h_x = &b(0);
      Vector<T> h_y(m);

      T * d_a;
      T * d_x;
      T * d_y;

      // Setup GPU memory
      CUDA_CALL(hipMalloc(&d_a, m * k * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_a, h_a, m * k * sizeof(T),
        hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_x, k * sizeof(T)));
      CUDA_CALL(hipMemcpy(d_x, h_x, k * sizeof(T),
          hipMemcpyHostToDevice));

      CUDA_CALL(hipMalloc(&d_y, m * sizeof(T)));
      CUDA_CALL(hipMemset(d_y, 0, m * sizeof(T)));

      // Launch kernel here
      dim3 dimBlock(BLOCK_SIZE);
      dim3 dimGrid(std::ceil((float)m / (BLOCK_SIZE)));

      CudaSharedMVKernel<<<dimGrid, dimBlock, BLOCK_SIZE * sizeof(T)>>>
        (d_a, d_x, d_y, m, k);
      // Device sync
      CUDA_CALL(hipDeviceSynchronize());

      // Transfer memories back, clear memrory, and return result
      CUDA_CALL(hipMemcpy(&h_y(0), d_y, m * sizeof(T),
        hipMemcpyDeviceToHost));
      CUDA_CALL(hipFree(d_a));
      CUDA_CALL(hipFree(d_x));
      CUDA_CALL(hipFree(d_y));

      return h_y;
    } else if (a.cols() != b.rows()) {
      std::cerr << "Matricies in gpu matrix multiply's sizes aren't compatible"
                << std::endl;
      exit(1);
    } else if (a.isZero() && b.isZero()) {
      std::cerr << "The maxtrix and the vector are empty"
                << std::endl;
      exit(1);
    } else if (a.isZero()) {
      std::cerr << "The maxtrix is empty"
                << std::endl;
      exit(1);
    } else if (b.isZero()) {
      std::cerr << "The vector is empty"
                << std::endl;
      exit(1);
    } else {
      std::cerr << "Unknown error"
                << std::endl;
      exit(1);
    }
  }
  template
  Vector<float> CudaSharedMVMultiply<float>::Multiply(const Matrix<float> &a, const Vector<float> &b);

}
